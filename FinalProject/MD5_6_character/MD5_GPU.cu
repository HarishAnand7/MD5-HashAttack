#include <math.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstring>

#define tmax 62
#define totalCombinations 56800235584 
#define numBlocks 57258303
#define threadsPerBlock  992

__constant__ uint32_t A = 0x67452301;
__constant__ uint32_t B = 0xefcdab89;
__constant__ uint32_t C = 0x98badcfe;
__constant__ uint32_t D = 0x10325476;

__constant__ uint32_t S[] = {7, 12, 17, 22, 7, 12, 17, 22, 7, 12, 17, 22, 7, 12, 17, 22,
                          5, 9, 14, 20, 5, 9, 14, 20, 5, 9, 14, 20, 5, 9, 14, 20,
                          4, 11, 16, 23, 4, 11, 16, 23, 4, 11, 16, 23, 4, 11, 16, 23,
                          6, 10, 15, 21, 6, 10, 15, 21, 6, 10, 15, 21, 6, 10, 15, 21};

__constant__ uint32_t K[] = {0xd76aa478, 0xe8c7b756, 0x242070db, 0xc1bdceee,
                       0xf57c0faf, 0x4787c62a, 0xa8304613, 0xfd469501,
                       0x698098d8, 0x8b44f7af, 0xffff5bb1, 0x895cd7be,
                       0x6b901122, 0xfd987193, 0xa679438e, 0x49b40821,
                       0xf61e2562, 0xc040b340, 0x265e5a51, 0xe9b6c7aa,
                       0xd62f105d, 0x02441453, 0xd8a1e681, 0xe7d3fbc8,
                       0x21e1cde6, 0xc33707d6, 0xf4d50d87, 0x455a14ed,
                       0xa9e3e905, 0xfcefa3f8, 0x676f02d9, 0x8d2a4c8a,
                       0xfffa3942, 0x8771f681, 0x6d9d6122, 0xfde5380c,
                       0xa4beea44, 0x4bdecfa9, 0xf6bb4b60, 0xbebfbc70,
                       0x289b7ec6, 0xeaa127fa, 0xd4ef3085, 0x04881d05,
                       0xd9d4d039, 0xe6db99e5, 0x1fa27cf8, 0xc4ac5665,
                       0xf4292244, 0x432aff97, 0xab9423a7, 0xfc93a039,
                       0x655b59c3, 0x8f0ccc92, 0xffeff47d, 0x85845dd1,
                       0x6fa87e4f, 0xfe2ce6e0, 0xa3014314, 0x4e0811a1,
                       0xf7537e82, 0xbd3af235, 0x2ad7d2bb, 0xeb86d391};

// Padding used to make the size (in bits) of the input congruent to 448 mod 512
__constant__ uint8_t PADDING[] = {0x80, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00,
                            0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00,
                            0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00,
                            0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00,
                            0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00,
                            0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00,
                            0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00,
                            0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00};


// Bit-manipulation functions defined by the MD5 algorithm
#define F(X, Y, Z) ((X & Y) | (~X & Z))
#define G(X, Y, Z) ((X & Z) | (Y & ~Z))
#define H(X, Y, Z) (X ^ Y ^ Z)
#define I(X, Y, Z) (Y ^ (X | ~Z))

// Rotates a 32-bit word left by n bits
__device__ uint32_t rotateLeft(uint32_t x, uint32_t n) 
{
    return (x << n) | (x >> (32 - n));
}
// MD5_Struct struct
struct MD5_Struct 
{
    uint64_t size = 0;
    uint32_t buffer[4] = {A, B, C, D};
    uint8_t input[64] = {0};
    uint8_t digest[16] = {0};
};


// Step on 512 bits of input with the main MD5 algorithm.
__device__ void MD5Step(uint32_t* buffer, uint32_t* input)
{
    uint32_t AA = buffer[0];
    uint32_t BB = buffer[1];
    uint32_t CC = buffer[2];
    uint32_t DD = buffer[3];

    uint32_t E;

    unsigned int j;

    for(unsigned int i = 0; i < 64; ++i){
        switch(i / 16){
            case 0:
                E = F(BB, CC, DD);
                j = i;
                break;
            case 1:
                E = G(BB, CC, DD);
                j = ((i * 5) + 1) % 16;
                break;
            case 2:
                E = H(BB, CC, DD);
                j = ((i * 3) + 5) % 16;
                break;
            default:
                E = I(BB, CC, DD);
                j = (i * 7) % 16;
                break;
        }

        uint32_t temp = DD;
        DD = CC;
        CC = BB;
        BB = BB + rotateLeft(AA + E + K[i] + input[j], S[i]);
        AA = temp;
    }

    buffer[0] += AA;
    buffer[1] += BB;
    buffer[2] += CC;
    buffer[3] += DD;
}

__device__ void MD5Init(MD5_Struct& content)
{
    content.size = 0;
    content.buffer[0] = A;
    content.buffer[1] = B;
    content.buffer[2] = C;
    content.buffer[3] = D;
}

__device__ void MD5Update(MD5_Struct& content, const uint8_t* input_buffer, size_t input_len)
{
    uint32_t input[16];
    unsigned int offset = static_cast<unsigned int>(content.size % 64);
    content.size += static_cast<uint64_t>(input_len);

    // Copy each byte in input_buffer into the next space in our context input
    for (size_t i = 0; i < input_len; ++i) 
    {
        content.input[offset++] = input_buffer[i];

        // If we've filled our context input, copy it into our local array input
        // then reset the offset to 0 and fill in a new buffer.
        // Every time we fill out a chunk, we run it through the algorithm
        // to enable some back and forth between CPU and I/O
        if (offset % 64 == 0) {
            for (size_t j = 0; j < 16; ++j) 
	    {
                // Convert to little-endian
                // The local variable `input` is our 512-bit chunk separated into 32-bit words
                // we can use in calculations
                input[j] = static_cast<uint32_t>(content.input[(j * 4) + 3]) << 24 |
                           static_cast<uint32_t>(content.input[(j * 4) + 2]) << 16 |
                           static_cast<uint32_t>(content.input[(j * 4) + 1]) << 8 |
                           static_cast<uint32_t>(content.input[(j * 4)]);
            }
            MD5Step(content.buffer, input);
            offset = 0;
        }
    }
}

__device__ void MD5Finalize(MD5_Struct& content) 
{
    uint32_t input[16];
    unsigned int offset = static_cast<unsigned int>(content.size % 64);
    unsigned int padding_length = (offset < 56) ? (56 - offset) : (120 - offset);

    // Fill in the padding and undo the changes to size that resulted from the update
    MD5Update(content, PADDING, padding_length);
    content.size -= static_cast<uint64_t>(padding_length);

    // Do a final update (internal to this function)
    // Last two 32-bit words are the two halves of the size (converted from bytes to bits)
    for (unsigned int j = 0; j < 14; ++j) 
    {
        input[j] = static_cast<uint32_t>(content.input[(j * 4) + 3]) << 24 |
                   static_cast<uint32_t>(content.input[(j * 4) + 2]) << 16 |
                   static_cast<uint32_t>(content.input[(j * 4) + 1]) << 8 |
                   static_cast<uint32_t>(content.input[(j * 4)]);
    }
    input[14] = static_cast<uint32_t>(content.size * 8);
    input[15] = static_cast<uint32_t>(content.size >> 32);

    MD5Step(content.buffer, input);

    // Move the result into digest (convert from little-endian)
    for (unsigned int i = 0; i < 4; ++i) {
        content.digest[(i * 4) + 0] = static_cast<uint8_t>(content.buffer[i] & 0x000000FF);
        content.digest[(i * 4) + 1] = static_cast<uint8_t>((content.buffer[i] & 0x0000FF00) >> 8);
        content.digest[(i * 4) + 2] = static_cast<uint8_t>((content.buffer[i] & 0x00FF0000) >> 16);
        content.digest[(i * 4) + 3] = static_cast<uint8_t>((content.buffer[i] & 0xFF000000) >> 24);
    }
}

__device__ void MD5String(const char *input, uint8_t *result)
 {
    MD5_Struct content;
    MD5Init(content);
    MD5Update(content, reinterpret_cast<const uint8_t*>(input), 6);
    MD5Finalize(content);
   
          result[0] = content.digest[0];
 	  result[1] = content.digest[1];
          result[2] = content.digest[2];
	  result[3] = content.digest[3];
          result[4] = content.digest[4];
	  result[5] = content.digest[5];
          result[6] = content.digest[6];
	  result[7] = content.digest[7];
          result[8] = content.digest[8];
	  result[9] = content.digest[9];
          result[10] = content.digest[10];
	  result[11] = content.digest[11];
          result[12] = content.digest[12];
	  result[13] = content.digest[13];
	  result[14] = content.digest[14];
	  result[15] = content.digest[15];
      
    
}
__device__ void FormatDigestToHex(const unsigned char *digest, char *output) 
{
    const char* hexChars = "0123456789abcdef";

    for (int i = 0; i < 16; ++i) 
    {
        output[i * 2] = hexChars[(digest[i] >> 4) & 0x0F];
        output[i * 2 + 1] = hexChars[digest[i] & 0x0F];
    }
    output[32] = '\0'; // Null-terminate the string
}


__global__ void MD5Kernel(const char *Input, char *targetHash) 
{
    unsigned char digest[16];
    MD5String(Input, digest);  // Assuming md5String computes the MD5 hash

    FormatDigestToHex(digest, targetHash);  // Convert digest to hex string
}

__constant__ char charset[] = "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789";
__global__ void bruteForceKernel(char *targetHash) 
{
    char password[7] = {0};  // 6 characters + null terminator
    unsigned char hash[16];  

    long long idx =(long long)  blockIdx.x * (long long)blockDim.x + threadIdx.x;
    if (idx >= totalCombinations) return;

    // Compute the n-th combination
    long long tmpIdx = idx;
    password[0] = charset[tmpIdx % tmax];
    tmpIdx /= tmax;
    password[1] = charset[tmpIdx % tmax];
    tmpIdx /= tmax;
    password[2] = charset[tmpIdx % tmax];
    tmpIdx /= tmax;
    password[3] = charset[tmpIdx % tmax];
    tmpIdx /= tmax;
    password[4] = charset[tmpIdx % tmax];
    tmpIdx /= tmax;
    password[5] = charset[tmpIdx % tmax];
    tmpIdx /= tmax;


    password[6]='\0';

    __shared__ bool match;
    match=true;


    char genhash[33];
    MD5String(password,hash);
    FormatDigestToHex(hash,genhash);

      if ((genhash[0] != targetHash[0])|(genhash[1] != targetHash[1])|(genhash[2] != targetHash[2])|(genhash[3] != targetHash[3])
  |(genhash[4] != targetHash[4]) |(genhash[5] != targetHash[5]) |(genhash[6] != targetHash[6]) | (genhash[7] != targetHash[7])
  | (genhash[8] != targetHash[8])| (genhash[9] != targetHash[9])  | (genhash[10] != targetHash[10])| (genhash[11] != targetHash[11])
  | (genhash[12] != targetHash[12])  | (genhash[13] != targetHash[13]) | (genhash[14] != targetHash[14]) | (genhash[15] != targetHash[15])
  | (genhash[16] != targetHash[16]) | (genhash[17] != targetHash[17])  | (genhash[18] != targetHash[18])| (genhash[19] != targetHash[19])
  | (genhash[20] != targetHash[20]) | (genhash[21] != targetHash[21]) | (genhash[22] != targetHash[22])  | (genhash[23] != targetHash[23])
  | (genhash[24] != targetHash[24]) | (genhash[25] != targetHash[25])| (genhash[26] != targetHash[26])  | (genhash[27] != targetHash[27])
  | (genhash[28] != targetHash[28]) | (genhash[39] != targetHash[29]) | (genhash[30] != targetHash[30]) | (genhash[31] != targetHash[31]))
      {  match =false;}


    if (match) 
    {
        printf("Password Cracked SMARTASSS !!!! :%s\n", password);
        return;
        	
    }
	
    

}




int main(int argc, char *argv[]) 
{
    const char* Input = argv[1];
    char* d_input;
    char* d_targetHash;
    char h_targetHash[33];
    char output[33];


    hipMallocManaged(&d_input, 7);
    hipMallocManaged(&d_targetHash, 33 * sizeof(char));
    hipMemcpy(d_input, Input, 7, hipMemcpyHostToDevice);

    // Launch the kernel
    MD5Kernel<<<1, 1>>>(d_input, d_targetHash);

    hipMemcpy(h_targetHash, d_targetHash, 33 * sizeof(char), hipMemcpyDeviceToHost);

    std::cout <<" Space Patrol Delta \n **************\n Enter your Bank Password: *******\n\n"; 

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    char* dev_targetHash;
    hipMallocManaged(&dev_targetHash, 33 * sizeof(char));
    hipMemcpy(dev_targetHash, h_targetHash, 33 * sizeof(char), hipMemcpyHostToDevice);

    //number of blocks = (totalCombinations + threadsPerBlock - 1) / threadsPerBlock;

    hipEventRecord(start, 0);

    bruteForceKernel<<<numBlocks, threadsPerBlock>>>(dev_targetHash);
	
    hipMemcpy(output,dev_targetHash,33 * sizeof(char),hipMemcpyDeviceToHost);	
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);


    // Calculate the elapsed time between start and stop
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "\nTotal time taken to crack the password :" << milliseconds << " milliseconds " <<std::endl;
    hipDeviceSynchronize();

    hipFree(d_input);
    hipFree(d_targetHash);
    hipFree(dev_targetHash);
   
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) 
    {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
        return -1;
    }

    return 0;
}



