

int main(int argc, char *argv[]) 
{
    const char* Input = argv[1];
    char* d_input;
    char* d_targetHash;
    char h_targetHash[32];
    char output[32];


    hipMalloc(&d_input, 7);
    hipMalloc(&d_targetHash, 32 * sizeof(char));
    hipMemcpy(d_input, Input, 7, hipMemcpyHostToDevice);

    // Launch the kernel
    md5Kernel<<<1, 1>>>(d_input, d_targetHash);

    hipMemcpy(h_targetHash, d_targetHash, 33 * sizeof(char), hipMemcpyDeviceToHost);

    std::cout <<" Space Patrol Delta \n **************\n Enter your Bank Password: *******\n\n"; 

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    char* dev_targetHash;
    hipMalloc(&dev_targetHash, 32 * sizeof(char));
    hipMemcpy(dev_targetHash, h_targetHash, 32 * sizeof(char), hipMemcpyHostToDevice);

    //number of blocks = (totalCombinations + threadsPerBlock - 1) / threadsPerBlock;

    hipEventRecord(start, 0);

    bruteForceKernel<<<numBlocks, threadsPerBlock>>>(dev_targetHash);
	
    hipMemcpy(output,dev_targetHash,33*sizeof(char),hipMemcpyDeviceToHost);	
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);


    // Calculate the elapsed time between start and stop
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "\nTotal time taken to crack the password :" << milliseconds << " milliseconds " <<std::endl;
    hipDeviceSynchronize();

    hipFree(d_input);
    hipFree(d_targetHash);
    hipFree(dev_targetHash);
   
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) 
    {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
        return -1;
    }

    return 0;
}


