#include "vscale.cu"
#include <iostream>
#include <random>
#include <chrono>
#include <cmath>
#include <hip/hip_runtime.h>

int main(int argc, char *argv[])
{

   int N = std::atoi(argv[1]);

   float *hA=new float[N];
   float *hB=new float[N];

   std::random_device rd;
   std::mt19937 gen(rd());
   std::uniform_real_distribution<float> dist_a(-10.0, 10.0);
   std::uniform_real_distribution<float> dist_b(0.0, 1.0);

    float *a, *b;

    int size= N*sizeof(float);

    hipMalloc((void**)&a, size);
    hipMalloc((void**)&b, size);

    for (int i = 0; i < N; i++)
    {
        hA[i] = dist_a(gen);
        hB[i] = dist_b(gen);
    }

    hipMemcpy(a,hA,size,hipMemcpyHostToDevice);
    hipMemcpy(b,hB,size,hipMemcpyHostToDevice);

    int blockSize = 512;
    int numBlocks = (N + blockSize - 1) / blockSize;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    vscale<<<numBlocks , blockSize>>>(a,b,N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float secs = 0.0;
    hipEventElapsedTime(&secs, start, stop);

    hipMemcpy(hB,b,size,hipMemcpyDeviceToHost);


    std::cout << secs << "\n";
    std::cout << hB[0] << "\n";
    std::cout << hB[N - 1] << "\n";


    hipError_t hipError_t = hipGetLastError();
      if (hipError_t != hipSuccess)
      {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(hipError_t));
        return 1;
      }

    hipDeviceSynchronize();

    hipFree(a);
    hipFree(b);

    delete[] hA;
    delete[] hB;

    return 0;
}
