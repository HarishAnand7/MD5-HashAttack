#include <stdio.h>
#include <hip/hip_runtime.h>
#include <random>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <iostream>
using namespace std;
using std::cout;

int main(int argc, char *argv[])
{
        int n = atoi(argv[1]);
        std::random_device rd;
        std::mt19937 gen(rd());
        std::uniform_real_distribution<float> dist1(-1.0f, 1.0f);


        thrust::host_vector<float> hA(n);
        for (int i = 0; i < n; i++)
        {
                hA[i] = dist1(gen);
        }
        thrust::device_vector<float> dA = hA;
        hipEvent_t start;
        hipEvent_t stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);
        float result = thrust::reduce(dA.begin() , dA.end(),0.0,thrust::plus<float>());
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipError_t hipError_t = hipGetLastError();
        if (hipError_t != hipSuccess)
        {
         fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(hipError_t));
         return 1;
        }

        float ms;
        hipEventElapsedTime(&ms, start, stop);
        std::cout << result << std::endl;
        std::cout << ms << std::endl;

        return 0;
}

